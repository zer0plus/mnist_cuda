#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../kernels.cuh"
#include "../consts.cuh"
#include <hipDNN.h>

void relu_cudnn(float* data, int size, hipdnnHandle_t cudnnHandle) {
    hipdnnActivationDescriptor_t activationDesc;
    hipdnnCreateActivationDescriptor(&activationDesc);
    hipdnnSetActivationDescriptor(activationDesc, 
                                HIPDNN_ACTIVATION_RELU,
                                HIPDNN_PROPAGATE_NAN, 
                                0.0);

    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc,
                            HIPDNN_TENSOR_NCHW,
                            HIPDNN_DATA_FLOAT,
                            1, 1, 1, size);

    float alpha = 1.0f;
    float beta = 0.0f;
    
    hipdnnActivationForward(cudnnHandle,
                        activationDesc,
                        &alpha,
                        tensorDesc,
                        data,
                        &beta,
                        tensorDesc,
                        data);

    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyTensorDescriptor(tensorDesc);
}

void relu_cpu(float *out, int size) {
    for (int i = 0; i < size; i++) {
        out[i] = out[i] > 0 ? out[i] : 0;
    }
}

void relu_derivative_cpu(float *grad, float *out, int size) {
    for (int i = 0; i < size; i++) {
        grad[i] *= out[i] > 0 ? 1 : 0;
    }
}

void initialize_data(float *data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = (float)(rand() % 20 - 10) / 2.0f;
    }
}

int compare_arrays(float *a, float *b, int size) {
    for (int i = 0; i < size; i++) {
        if (fabs(a[i] - b[i]) > 1e-5) {
            return 0;
        }
    }
    return 1;
}

void run_test(int size) {
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // Allocate host memory
    float *h_input = (float*)malloc(size * sizeof(float));
    float *h_output_cpu = (float*)malloc(size * sizeof(float));
    float *h_output_gpu = (float*)malloc(size * sizeof(float));
    float *h_grad_cpu = (float*)malloc(size * sizeof(float));
    float *h_grad_gpu = (float*)malloc(size * sizeof(float));
    float *h_grad_deriv_cpu = (float*)malloc(size * sizeof(float));
    float *h_grad_deriv_gpu = (float*)malloc(size * sizeof(float));

    // Initialize data
    initialize_data(h_input, size);
    initialize_data(h_grad_cpu, size);
    memcpy(h_grad_deriv_cpu, h_grad_cpu, size * sizeof(float));

    // Allocate device memory
    float *d_input, *d_output, *d_grad;
    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));
    hipMalloc((void**)&d_grad, size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grad, h_grad_cpu, size * sizeof(float), hipMemcpyHostToDevice);

    // CPU Forward Pass
    memcpy(h_output_cpu, h_input, size * sizeof(float));
    hipEvent_t start, stop;
    float cpu_time, gpu_time;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    relu_cpu(h_output_cpu, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_time, start, stop);

    // GPU Forward Pass
    hipMemcpy(d_output, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(start);
    relu_cuda(d_output, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    hipMemcpy(h_output_gpu, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Forward Pass Validation
    int forward_valid = compare_arrays(h_output_cpu, h_output_gpu, size);

    // CPU Derivative Pass
    float cpu_deriv_time;
    hipEventRecord(start);
    relu_derivative_cpu(h_grad_deriv_cpu, h_output_cpu, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_deriv_time, start, stop);

    // GPU Derivative Pass
    float gpu_deriv_time;
    const int block_size = 256;
    const int num_blocks = (size + block_size - 1) / block_size;
    
    hipEventRecord(start);
    optimized_relu_derivative_kernel<<<num_blocks, block_size>>>(d_grad, d_output, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_deriv_time, start, stop);
    hipMemcpy(h_grad_deriv_gpu, d_grad, size * sizeof(float), hipMemcpyDeviceToHost);

    // Derivative Validation
    int deriv_valid = compare_arrays(h_grad_deriv_cpu, h_grad_deriv_gpu, size);

    // Structured Output
    printf("RELU_RESULTS:\n");
    printf("InputSize: %d\n", size);
    printf("Forward_CPU_Time: %.4f\n", cpu_time);
    printf("Forward_GPU_Time: %.4f\n", gpu_time);
    printf("Forward_Speedup: %.2f\n", cpu_time / gpu_time);
    printf("Forward_Valid: %d\n", forward_valid);
    printf("Derivative_CPU_Time: %.4f\n", cpu_deriv_time);
    printf("Derivative_GPU_Time: %.4f\n", gpu_deriv_time);
    printf("Derivative_Speedup: %.2f\n", cpu_deriv_time / gpu_deriv_time);
    printf("Derivative_Valid: %d\n", deriv_valid);
    printf("END_RESULTS\n");

    // Cleanup
    free(h_input);
    free(h_output_cpu);
    free(h_output_gpu);
    free(h_grad_cpu);
    free(h_grad_gpu);
    free(h_grad_deriv_cpu);
    free(h_grad_deriv_gpu);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_grad);
    hipdnnDestroy(cudnnHandle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char** argv) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input_size>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);
    if (size <= 0) {
        fprintf(stderr, "Invalid input size: %d\n", size);
        return 1;
    }

    run_test(size);
    return 0;
}