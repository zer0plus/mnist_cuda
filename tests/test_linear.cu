#include "hip/hip_runtime.h"
#include "../kernels.cuh"
#include "../mnist.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>

#define HIDDEN_LAYER_TEST_SIZE (HIDDEN_LAYER_SIZE * 100)
#define OUT_LAYER_TEST_SIZE (OUTPUT_LAYER_SIZE * 100)

// Debug code to print matrices
void print_matrices(float *inp, float *weights, float *out, int in_size, int out_size) {
    printf("Input vector:\n");
    for (int i = 0; i < min(10, in_size); i++) {
        printf("%.4f ", inp[i]);
    }
    printf("\n\n");
    
    printf("Weights matrix (row-major layout):\n");
    for (int i = 0; i < min(5, in_size); i++) {
        for (int j = 0; j < min(5, out_size); j++) {
            printf("%.4f ", weights[i * out_size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_ele(float* cpu_ele, float* gpu_ele, size_t size) {
    // printf("CPU ele: ");
    // for (int i = 0; i < 20; i++) {
    //     printf("%f ", cpu_ele[i]);
    // }
    // printf("\n");
    
    // printf("GPU ele: ");
    // for (int i = 0; i < 20; i++) {
    //     printf("%f ", gpu_ele[i]);
    // }
    // printf("\n");

    for (int i = 0; i < 100; i++) {
        if (fabs(cpu_ele[i] - gpu_ele[i]) > 1e-5) {
            printf("Mismatch at index %d: CPU = %f, GPU = %f\n", i, cpu_ele[i], gpu_ele[i]);
        }
    }
    // printf("Last element at index %zd: CPU = %f, GPU = %f\n", size-1, cpu_ele[size-1], gpu_ele[size-1]);
    printf("Last 10 elements check\n");

    for (int i = size-1; size-11 < i; i--) {
        if (fabs(cpu_ele[i] - gpu_ele[i]) > 1e-5) {
            printf("Mismatch at index %d: CPU = %f, GPU = %f\n", i, cpu_ele[i], gpu_ele[i]);
        }
    }
}

int compare_arrays(float *a, float *b, size_t size) {
    for (size_t i = 0; i < size; i++) {
        if (abs(a[i] - b[i]) > 1e-4) {
            return 0;
        }
    }
    return 1;
}

void initialize_data(float *data, size_t size) {
    for (size_t i = 0; i < size; i++) {
        data[i] = (float)rand() / RAND_MAX; // Random values between 0 and 1
    }
}


#ifdef RUN_LINEAR_TEST
int main() {
    // create dummy linear layers with different in and out test sizes
    // init cpu layer with data and copy it to gpu to make data same
    init_cublas();

    srand(time(NULL));
    printf("\nLinear Test with data size: %d \n", HIDDEN_LAYER_TEST_SIZE);
    float *h_out_cpu, *h_hidden_cpu, *h_out_cuda, *h_hidden_cuda, *h_input;
    CUDA_CHECK(hipHostMalloc((void **) &h_input, HIDDEN_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipHostMalloc((void **) &h_out_cuda, OUT_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipHostMalloc((void **) &h_out_cpu, OUT_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipHostMalloc((void **) &h_hidden_cuda, HIDDEN_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipHostMalloc((void **) &h_hidden_cpu, HIDDEN_LAYER_TEST_SIZE * sizeof(float)));

    float *d_input, *d_out, *d_hidden;
    CUDA_CHECK(hipMalloc((void **) &d_input, HIDDEN_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &d_out, OUT_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &d_hidden, HIDDEN_LAYER_TEST_SIZE * sizeof(float)));

    GenericLayer cpu_hidden;
    GenericLayer gpu_hidden;

    initialize_data(h_input, HIDDEN_LAYER_TEST_SIZE);
    init_layer(&cpu_hidden, HIDDEN_LAYER_TEST_SIZE, OUT_LAYER_TEST_SIZE);
    gpu_hidden.in_size = cpu_hidden.in_size;
    gpu_hidden.out_size = cpu_hidden.out_size;
    size_t num_layer_elements = cpu_hidden.in_size * cpu_hidden.out_size;
    
    CUDA_CHECK(hipMemcpy(d_input, h_input, HIDDEN_LAYER_TEST_SIZE * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void **) &gpu_hidden.weights, num_layer_elements * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &gpu_hidden.biases, OUT_LAYER_TEST_SIZE * sizeof(float)));
    CUDA_CHECK(hipMemcpy(gpu_hidden.weights, cpu_hidden.weights, num_layer_elements * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(gpu_hidden.biases, cpu_hidden.biases, OUT_LAYER_TEST_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Print CPU hidden layer weights and biases
    printf("CPU Hidden Layer Weights:\n");
    for (int i = 0; i < 20; i++) {
        printf("%.4f ", cpu_hidden.weights[i]);
    }
    printf("\n");
    printf("CPU Hidden Layer Biases:\n");
    for (int i = 0; i < 10; i++) {
        printf("%.4f ", cpu_hidden.biases[i]);
    }
    printf("\n");

    // Print GPU hidden layer weights and biases
    print_device_tensor("GPU Hidden Layer Weights", gpu_hidden.weights, num_layer_elements, 20);
    print_device_tensor("GPU Hidden Layer Biases", gpu_hidden.biases, OUT_LAYER_TEST_SIZE, 10);

    hipEvent_t start_cpu, stop_cpu;
    float cpu_time;
    hipEventCreate(&start_cpu);
    hipEventCreate(&stop_cpu);
    hipEventRecord(start_cpu);
    linear(&cpu_hidden, h_input, h_out_cpu);
    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);
    hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    print_matrices(h_input, cpu_hidden.weights, h_out_cpu, HIDDEN_LAYER_TEST_SIZE, OUT_LAYER_TEST_SIZE);

    hipEvent_t start_gpu, stop_gpu;
    float gpu_time;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);
    linear_cuda_cublas(&gpu_hidden, d_input, d_out);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    CUDA_CHECK(hipMemcpy(h_out_cuda, d_out, OUT_LAYER_TEST_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    int result = compare_arrays(h_out_cpu, h_out_cuda, OUT_LAYER_TEST_SIZE);
    if (result) {
        printf("\nLinear Test Passed: CPU and CUDA results match.\n");
    } else {
        printf("\nLinear Test Failed: CPU and CUDA results do not match.\n");
    }
   
       // Before running the linear operation
    print_device_tensor("GPU Input", d_input, HIDDEN_LAYER_TEST_SIZE, 20);
    // After running the linear operation
    print_device_tensor("GPU Output", d_out, OUT_LAYER_TEST_SIZE, 20);

    print_ele(h_out_cpu, h_out_cuda, OUT_LAYER_TEST_SIZE);
    printf(" CPU time: %f ms\n", cpu_time);
    printf(" GPU time: %f ms\n", gpu_time);
    printf(" Speedup: %fx\n", cpu_time / gpu_time);

    hipHostFree(h_input);
    hipHostFree(h_out_cuda);
    hipHostFree(h_out_cpu);
    hipHostFree(h_hidden_cuda);
    hipHostFree(h_hidden_cpu);
    hipFree(d_input);
    hipFree(d_out);
    hipFree(d_hidden);
}
#endif