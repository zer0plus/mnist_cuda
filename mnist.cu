#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include "consts.cuh"
#include <hip/hip_runtime.h>
// The MNIST image/label file structure is as follows:

//     [offset] [type] [value] [description]
//     0000 32 bit int 0x00000803 magic number
//     0004 32 bit int 60000 number of images
//     0008 32 bit int 28 number of rows
//     0012 32 bit int 28 number of columns
//     0016 unsigned byte ?? pixel
//     0017 unsigned byte ?? pixel
//     ........
//     xxxx unsigned byte ?? pixel

void print_device_tensor(const char* tensor_name, float* d_ptr, int shape_size, int num_elements_to_print) {
    // Get the actual allocated size using CUDA Driver API
    hipDeviceptr_t base;
    size_t actual_size;
    hipMemGetAddressRange(&base, &actual_size, (hipDeviceptr_t)d_ptr);
    size_t actual_elements = actual_size / sizeof(float);
    
    // Ensure we don't try to print more elements than exist
    num_elements_to_print = (num_elements_to_print > shape_size) ? shape_size : num_elements_to_print;
    
    // Allocate host memory for the elements we want to print
    float* h_data = (float*)malloc(num_elements_to_print * sizeof(float));
    hipMemcpy(h_data, d_ptr, num_elements_to_print * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print tensor information with both logical shape and actual elements
    printf("%s: dtype=float32, shape=(%d,), allocated_elements=%zu, size_in_bytes=%zu\n", 
           tensor_name, shape_size, actual_elements, actual_size);
    
    // Print elements
    printf("First %d elements: [", num_elements_to_print);
    for (int i = 0; i < num_elements_to_print; i++) {
        printf("%.4f", h_data[i]);
        if (i < num_elements_to_print - 1) {
            printf(", ");
        }
    }
    printf("]%s\n", shape_size > num_elements_to_print ? ", ...]" : "]");
    
    free(h_data);
}




// Kernel to initialize random number generators
__global__ void init_curand_states(hiprandState *states, unsigned long seed, size_t total_weights) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_weights) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void init_weights_kernel(float *weights, size_t total_weights, float scale, hiprandState *states) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_weights) {
        hiprandState localState = states[idx];
        weights[idx] = (hiprand_uniform(&localState) - 0.5f) * 2 * scale;
        states[idx] = localState;
    }
}

void init_layer_cuda(GenericLayer *layer, int in_size, int out_size) {
    size_t total_weights = in_size * out_size;
    float scale = sqrtf(2.0f / in_size);
    printf("CUDA - Total weights: %zu, Scale: %f\n", total_weights, scale);

    layer->in_size = in_size;
    layer->out_size = out_size;

    CUDA_CHECK(hipMalloc((void **)&layer->weights, total_weights * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&layer->biases, out_size * sizeof(float)));

    size_t block_size = 256;
    size_t num_blocks = (total_weights + block_size - 1) / block_size;

    hiprandState *d_states;
    CUDA_CHECK(hipMalloc(&d_states, total_weights * sizeof(hiprandState)));

    unsigned long seed = time(NULL);
    printf("Launching kernels with %zu blocks of %zu threads\n", num_blocks, block_size);

    init_curand_states<<<num_blocks, block_size>>>(d_states, seed, total_weights);
    CUDA_CHECK(hipGetLastError());
    
    init_weights_kernel<<<num_blocks, block_size>>>(layer->weights, total_weights, scale, d_states);
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemset(layer->biases, 0, out_size * sizeof(float)));
    CUDA_CHECK(hipFree(d_states));
}


__global__ void normalize_imgs_kernel(unsigned char* raw_imgs, float* d_normalized_imgs, int total_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        d_normalized_imgs[idx] = raw_imgs[idx] / 255.0f;
    }
}

void read_mnist_imgs_cuda(const char *filename, float **d_imgs, int *num_imgs) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        exit(1);
    }

    int tmp, rows, cols;
    size_t read_elements;

    read_elements = fread(&tmp, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read magic number\n");
        fclose(file);
        exit(1);
    }

    read_elements = fread(num_imgs, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read number of images\n");
        fclose(file);
        exit(1);
    }
    *num_imgs = __builtin_bswap32(*num_imgs);

    read_elements = fread(&rows, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read number of rows\n");
        fclose(file);
        exit(1);
    }
    rows = __builtin_bswap32(rows);

    read_elements = fread(&cols, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read number of columns\n");
        fclose(file);
        exit(1);
    }
    cols = __builtin_bswap32(cols);

    int total_pixels = (*num_imgs) * IMAGE_H * IMAGE_W;
    unsigned char *h_imgs = (unsigned char *)malloc(total_pixels);

    read_elements = fread(h_imgs, sizeof(unsigned char), total_pixels, file);
    if (read_elements != total_pixels) {
        fprintf(stderr, "Error: Failed to read image data\n");
        free(h_imgs);
        fclose(file);
        exit(1);
    }

    fclose(file);

    // Allocate device memory for raw and normalized images
    // float *d_imgs;
    unsigned char *d_raw_imgs;
    CUDA_CHECK(hipMalloc((void **)&d_raw_imgs, total_pixels * sizeof(unsigned char)));

    CUDA_CHECK(hipMalloc((void **)d_imgs, total_pixels * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_raw_imgs, h_imgs, total_pixels * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    int block_size = 256;
    int num_blocks = (total_pixels + block_size - 1) / block_size;
    normalize_imgs_kernel<<<num_blocks, block_size>>>(d_raw_imgs, *d_imgs, total_pixels);
    CUDA_CHECK(hipGetLastError());

    free(h_imgs);
    CUDA_CHECK(hipFree(d_raw_imgs));
}

void read_mnist_labels_cuda(const char *filename, unsigned char **d_labels, int *num_labels) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        exit(1);
    }

    int tmp;
    size_t read_elements;

    read_elements = fread(&tmp, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read magic number\n");
        fclose(file);
        exit(1);
    }

    read_elements = fread(num_labels, sizeof(int), 1, file);
    if (read_elements != 1) {
        fprintf(stderr, "Error: Failed to read number of labels\n");
        fclose(file);
        exit(1);
    }
    *num_labels = __builtin_bswap32(*num_labels);

    unsigned char *h_labels = (unsigned char *)malloc(*num_labels);

    read_elements = fread(h_labels, sizeof(unsigned char), (*num_labels), file);
    if (read_elements != (*num_labels)) {
        fprintf(stderr, "Error: Failed to read label data\n");
        free(h_labels);
        fclose(file);
        exit(1);
    }
    fclose(file);

    CUDA_CHECK(hipMalloc((void **)d_labels, *num_labels * sizeof(unsigned char)));
    CUDA_CHECK(hipMemcpy(*d_labels, h_labels, *num_labels * sizeof(unsigned char), hipMemcpyHostToDevice));

    free(h_labels);
}

__global__ void swap_pixels_kernel(float *imgs, int i, int j, int img_size) {
    int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixel_idx < img_size) {
        float tmp = imgs[i * img_size + pixel_idx];
        imgs[i * img_size + pixel_idx] = imgs[j * img_size + pixel_idx];
        imgs[j * img_size + pixel_idx] = tmp;
    }
}


__global__ void shuffle_kernel(float *imgs, unsigned char *labels, int *shuffled_indices, int num_imgs, int img_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_imgs) {
        int j = shuffled_indices[idx];
        
        // Swap labels
        unsigned char tmp_label = labels[idx];
        labels[idx] = labels[j];
        labels[j] = tmp_label;
        
        int block_size = 256;
        int num_blocks = (img_size + block_size - 1) / block_size;

        swap_pixels_kernel<<<num_blocks, block_size>>>(imgs, idx, j, img_size);
        // CUDA_CHECK(hipGetLastError());
    }
}

__global__ void shuffle_naive_kernel(float *imgs, unsigned char *labels, int *shuffled_indices, int num_imgs, int img_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_imgs) {
        int j = shuffled_indices[idx];
        
        // Swap labels
        unsigned char tmp_label = labels[idx];
        labels[idx] = labels[j];
        labels[j] = tmp_label;
        
        // Swap pixels
        for (int pixel_idx = 0; pixel_idx < img_size; pixel_idx++) {
            float tmp_pixel = imgs[idx * img_size + pixel_idx];
            imgs[idx * img_size + pixel_idx] = imgs[j * img_size + pixel_idx];
            imgs[j * img_size + pixel_idx] = tmp_pixel;
        }
    }
}

void shuffle_data_cuda(float *imgs, unsigned char *labels, int num_imgs, int img_size) {
    int *h_indices = (int *)malloc(num_imgs * sizeof(int));
    for (int i = 0; i < num_imgs; i++) {
        h_indices[i] = i;
    }
    for (int i = num_imgs-1; i > 0; i--) {
        int j = rand() % (i+1);
        int temp = h_indices[i];
        h_indices[i] = h_indices[j];
        h_indices[j] = temp;
    }

    int *d_indices;
    CUDA_CHECK(hipMalloc(&d_indices, num_imgs * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_indices, h_indices, num_imgs * sizeof(int), hipMemcpyHostToDevice));
    
    int block_size = 256;
    int num_blocks = (num_imgs + block_size - 1) / block_size;
    // shuffle_kernel<<<num_blocks, block_size>>>(imgs, labels, d_indices, num_imgs, img_size);
    shuffle_naive_kernel<<<num_blocks, block_size>>>(imgs, labels, d_indices, num_imgs, img_size);
    CUDA_CHECK(hipGetLastError());

    // Synchronize to ensure all pixel swaps are complete
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_indices));
    free(h_indices);
}

__global__ void copy_biases_kernel(float *biases, float *out, int out_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < out_size) {
        out[idx] = biases[idx];
    }
}

__global__ void dot_product_kernel(float *weights, float *inp, float *partial_sums, int in_size, int out_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < in_size && idy < out_size) {
        atomicAdd(&partial_sums[idy], inp[idx] * weights[idx * out_size + idy]);
    }
}

__global__ void matrix_multiply_kernel(float *weights, float *inp, float *out, float *partial_sums, int in_size, int out_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < out_size) {
        out[idx] += partial_sums[idx];
    }
}

void linear_cuda(GenericLayer *layer, float *inp, float *out) {
    int block_size = 256;
    int num_blocks = (layer->out_size + block_size - 1) / block_size;
    copy_biases_kernel<<<num_blocks, block_size>>>(layer->biases, out, layer->out_size);
    CUDA_CHECK(hipGetLastError());
    // DO YOU REALLY NEED TO CALL SYNC THIS EARLY, CHECK IT
    CUDA_CHECK(hipDeviceSynchronize());

    static int count = 0;
    if (count == 0) {
        // Debug print for bias initialization
        print_device_tensor("\nBias tensor copy: ", out, layer->out_size, 12);
        count++;
    }

    float *d_partial_sums;
    CUDA_CHECK(hipMalloc(&d_partial_sums, layer->out_size * sizeof(float)));
    CUDA_CHECK(hipMemset(d_partial_sums, 0, layer->out_size * sizeof(float)));

    dim3 block_size_dot(16, 16);
    dim3 grid_size_dot((layer->in_size + block_size_dot.x - 1) / block_size_dot.x,
                       (layer->out_size + block_size_dot.y - 1) / block_size_dot.y);

    // dim3 threads_per_block(16, 16);
    // dim3 num_blocks(
    //     (layer->in_size + threads_per_block.x - 1) / threads_per_block.x,
    //     (layer->out_size + threads_per_block.y - 1) / threads_per_block.y
    // );

    dot_product_kernel<<<grid_size_dot, block_size_dot>>>(layer->weights, inp, d_partial_sums, layer->in_size, layer->out_size);
    CUDA_CHECK(hipGetLastError());

    int num_blocks_multiply = (layer->out_size + block_size - 1) / block_size;
    matrix_multiply_kernel<<<num_blocks_multiply, block_size>>>(layer->weights, inp, out, d_partial_sums, layer->in_size, layer->out_size);
    CUDA_CHECK(hipGetLastError());

    if (count == 1) {
        // Debug print for bias initialization
        print_device_tensor("\nout after linear:", out, layer->out_size, 12);
        count++;
    }

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_partial_sums));
}

__global__ void exp_subtract_sum_kernel(float *inp, int size, float *max_val, float *sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inp[idx] = expf(inp[idx] - *max_val);
        atomicAdd(sum, inp[idx]);
    }
}

__global__ void divide_kernel(float *inp, int size, float *sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        inp[idx] /= *sum;
    }
}

void softmax_cuda(float *d_inp, int size) {
    float *d_max, *d_sum;

    CUDA_CHECK(hipMalloc(&d_max, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum, sizeof(float)));

    // Find max value
    thrust::device_ptr<float> dev_ptr(d_inp);
    thrust::device_ptr<float> max_ptr = thrust::max_element(dev_ptr, dev_ptr + size);
    CUDA_CHECK(hipMemcpy(d_max, thrust::raw_pointer_cast(max_ptr), sizeof(float), hipMemcpyDeviceToDevice));

    CUDA_CHECK(hipMemset(d_sum, 0, sizeof(float)));
    
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    exp_subtract_sum_kernel<<<num_blocks, block_size>>>(d_inp, size, d_max, d_sum);
    CUDA_CHECK(hipGetLastError());
    
    divide_kernel<<<num_blocks, block_size>>>(d_inp, size, d_sum);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_sum));
    CUDA_CHECK(hipFree(d_max));
}

__global__ void relu_kernel(float *out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        out[idx] = out[idx] > 0 ? out[idx] : 0;
    }
}

__global__ void relu_derivative_kernel(float *grad, float *out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        grad[idx] *= out[idx] > 0 ? 1 : 0;
    }
}

__global__ void backward_kernel(float *weights, float *biases, float *inp, float *out_grad, float *in_grad, int in_size, int out_size, float lr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < in_size) {
        float *weight_row_start = &weights[idx * out_size];
        float input_i = inp[idx];
        
        if (in_grad) {
            float temp_grad = 0.0f;
            for (int j = 0; j < out_size; j++) {
                temp_grad += out_grad[j] * weight_row_start[j];
                weight_row_start[j] -= lr * (out_grad[j] * input_i);
            }
            atomicAdd(&in_grad[idx], temp_grad);
        }
        else {
            for (int j = 0; j < out_size; j++) {
                weight_row_start[j] -= lr * (out_grad[j] * input_i);
            }
        }
    }
    
    if (idx < out_size) {
        biases[idx] -= lr * out_grad[idx];
    }
}

void backward_cuda(GenericLayer *layer, float *inp, float *out_grad, float *in_grad, float lr) {
    int block_size = 256;
    int num_blocks_in = (layer->in_size + block_size - 1) / block_size;
    int num_blocks_out = (layer->out_size + block_size - 1) / block_size;
    
    int num_blocks = max(num_blocks_in, num_blocks_out);
    
    backward_kernel<<<num_blocks, block_size>>>(layer->weights, layer->biases, inp, out_grad, in_grad, layer->in_size, layer->out_size, lr);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void update_out_grad_kernel(float *out_grad, float *output, unsigned char *d_labels, int d_label_idx) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < OUTPUT_LAYER_SIZE) {
        out_grad[idx] = output[idx] - (idx == d_labels[d_label_idx]);
    }
}



float* train_mnist_cuda(Network *net, float *inp, unsigned char *d_labels, int d_label_idx, float lr) {
    static float *d_final_output, *d_hidden_out, *d_out_grad, *d_hidden_grad;
    static bool first_run = true;

    if (first_run) {
        CUDA_CHECK(hipMalloc(&d_final_output, OUTPUT_LAYER_SIZE * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_hidden_out, HIDDEN_LAYER_SIZE * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_out_grad, OUTPUT_LAYER_SIZE * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_hidden_grad, HIDDEN_LAYER_SIZE * sizeof(float)));
        first_run = false;
    }

    // ZERO GRAD
    CUDA_CHECK(hipMemset(d_out_grad, 0, OUTPUT_LAYER_SIZE * sizeof(float)));
    CUDA_CHECK(hipMemset(d_hidden_grad, 0, HIDDEN_LAYER_SIZE * sizeof(float)));

    // Inp to Hidden layer fwd pass
    linear_cuda(&net->hidden, inp, d_hidden_out);

    int block_size = 256;
    int num_blocks_hidden = (HIDDEN_LAYER_SIZE + block_size - 1) / block_size;
    relu_kernel<<<num_blocks_hidden, block_size>>>(d_hidden_out, HIDDEN_LAYER_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Hidden to out layer fwd pass
    linear_cuda(&net->output, d_hidden_out, d_final_output);
    softmax_cuda(d_final_output, OUTPUT_LAYER_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    // Compute out gradient
    int num_blocks_out = (OUTPUT_LAYER_SIZE + block_size - 1) / block_size;
    update_out_grad_kernel<<<num_blocks_out, block_size>>>(d_out_grad, d_final_output, d_labels, d_label_idx);
    CUDA_CHECK(hipGetLastError());

    // output to hidden layer bwd pass
    backward_cuda(&net->output, d_hidden_out, d_out_grad, d_hidden_grad, lr);

    // Backprop through ReLU(derivative) Activation
    relu_derivative_kernel<<<num_blocks_hidden, block_size>>>(d_hidden_grad, d_hidden_out, HIDDEN_LAYER_SIZE);
    CUDA_CHECK(hipGetLastError());

    // hidden to output layer bwd pass
    backward_cuda(&net->hidden, inp, d_hidden_grad, NULL, lr);
    
    CUDA_CHECK(hipDeviceSynchronize());

    float *final_output = (float *)malloc(OUTPUT_LAYER_SIZE * sizeof(float));
    CUDA_CHECK(hipMemcpy(final_output, d_final_output, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    
    return final_output;
}

// forward only 
int forward(Network *net, float *inp) {
    static float *d_hidden_out, *d_final_output;
    static bool first_run = true;

    if (first_run) {
        CUDA_CHECK(hipMalloc(&d_final_output, OUTPUT_LAYER_SIZE * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_hidden_out, HIDDEN_LAYER_SIZE * sizeof(float)));
        first_run = false;
    }
    linear_cuda(&net->hidden, inp, d_hidden_out);

    int block_size = 256;
    int num_blocks = (HIDDEN_LAYER_SIZE + block_size - 1) / block_size;
    relu_kernel<<<num_blocks, block_size>>>(d_hidden_out, HIDDEN_LAYER_SIZE);
    CUDA_CHECK(hipGetLastError());
    
    linear_cuda(&net->output, d_hidden_out, d_final_output);
    softmax_cuda(d_final_output, OUTPUT_LAYER_SIZE);

    float* final_out = (float *)malloc(OUTPUT_LAYER_SIZE * sizeof(float));
    CUDA_CHECK(hipMemcpy(final_out, d_final_output, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    
    int ans = 0;
    // gettin the max probability class from the softmax as ans
    for (int i = 0; i < OUTPUT_LAYER_SIZE; i++) {
        if (final_out[i] > final_out[ans]) {
            ans = i;
        }
    }
    free(final_out);
    return ans;
}

#ifdef RUN_MNIST_CUDA
int main() {
    Network mnist_net;
    InputData_GPU data = {0};
    float lr = LEARNING_RATE;
    clock_t start, end;
    double gpu_time_used;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Assume using device 0
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Shared memory per block: %zu bytes\n", prop.sharedMemPerBlock);
    printf("Warp size: %d\n", prop.warpSize);

    srand(time(NULL));

    init_layer_cuda(&mnist_net.hidden, INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE);
    init_layer_cuda(&mnist_net.output, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);

    // UP UNTIL HERE IT WORKS!

    // imgs & labels are directly loaded to device
    read_mnist_imgs_cuda(TRAIN_IMG_PATH, &data.imgs, &data.num_imgs);
    read_mnist_labels_cuda(TRAIN_LABEL_PATH, &data.labels, &data.num_imgs);

    // shuffle_data_cuda(data.imgs, data.labels, data.num_imgs, IMAGE_SIZE);

    int train_size = (int)(data.num_imgs * TRAIN_SPLIT);
    int test_size = data.num_imgs - train_size;
    unsigned char* h_labels = (unsigned char*) malloc(data.num_imgs * sizeof(unsigned char));
    // float* h_imgs = (float*) malloc(data.num_imgs * INPUT_LAYER_SIZE * sizeof(float));
    float *final_out;
    float total_loss = 0;

    CUDA_CHECK(hipMemcpy(h_labels, data.labels, data.num_imgs * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpy(h_imgs, data.imgs, data.num_imgs * INPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        start = clock();
        total_loss = 0;
        for (int i = 0; i < train_size; i++) {
            float *current_img = data.imgs + (i * IMAGE_SIZE);
            if (i == 0 and epoch == 0) {
                printf("Image at iteration 0:\n");
                float host_img[IMAGE_SIZE];
                CUDA_CHECK(hipMemcpy(host_img, current_img, IMAGE_SIZE * sizeof(float), hipMemcpyDeviceToHost));
                for (int row = 0; row < 28; row++) {
                    for (int col = 0; col < 28; col++) {
                        if (host_img[row * 28 + col] > 0.0f) {
                            printf("X");
                        } else {
                            printf(" ");
                        }
                    }
                    printf("\n");
                }
                printf("\n");
            }
            final_out = train_mnist_cuda(&mnist_net, current_img, data.labels, i, lr);
            // printf("CURR LABEL: %u\n", h_labels[i]);
            total_loss -= logf(fmaxf(final_out[h_labels[i]], 1e-10f));
        }

        int correct = 0;
        for (int i = train_size; i < data.num_imgs; i++) {
            float *test_img = data.imgs + (i * IMAGE_SIZE);
            if (forward(&mnist_net, test_img) == h_labels[i]) {
                correct++;
            }
        }
        end = clock();
        gpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

        printf("Epoch %d, Accuracy: %.2f%%, Avg Loss: %.4f, Time: %.2f seconds\n", 
            epoch + 1, (float)correct / test_size * 100, total_loss / train_size, gpu_time_used);
    }

    CUDA_CHECK(hipFree(mnist_net.hidden.weights));
    CUDA_CHECK(hipFree(mnist_net.hidden.biases));
    CUDA_CHECK(hipFree(mnist_net.output.weights));
    CUDA_CHECK(hipFree(mnist_net.output.biases));
    CUDA_CHECK(hipFree(data.imgs));
    CUDA_CHECK(hipFree(data.labels));
    free(final_out);
    return 0;
}
#endif